#include "hip/hip_runtime.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include <>
using namespace std;

__global__ void Hello()
{
	printf("Hello from GPU %d \n", threadIdx.x);

}

int main()
{
	cout << "Hello from CPU" << endl;
	Hello << <3, 2 >> > ();

	hipDeviceSynchronize();
	return 0;

}
