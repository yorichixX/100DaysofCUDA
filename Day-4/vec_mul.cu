#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include <>
using namespace std;

__global__ void vecMul(float* A, float* B, float* C, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n)
	{
		C[index] = A[index] * B[index];
	}

}

int main()
{
	int n = 600;
	size_t size = n * sizeof(float);

	float *A_h, *B_h, *C_h;

	 A_h = new float[n];
	 B_h = new float[n];
	 C_h = new float[n];

	for (int i = 0;i <= n-1;i++)
	{
		A_h[i] = i * 2;
		B_h[i] = i * 4;
	}

	float* A_d, * B_d, * C_d;
	hipMalloc(&A_d, size);
	hipMalloc(&B_d, size);
	hipMalloc(&C_d, size);

	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
	int block = (n - 256 + 1) / 256;
	vecMul << <block, 256 >> > (A_d, B_d, C_d, n);
	hipDeviceSynchronize();
	hipMemcpy(C_h, C_d,size, hipMemcpyDeviceToHost);
	

	for (int i = 0; i < 50; ++i)  //outputting just 50 elements from the resultant vector.
	{
		if (i % 10 == 0)
		{
			cout << endl;
		}
		cout << i+1 << ". " << C_h[i] << endl;
	}

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	delete[] A_h;
	delete[] B_h;
	delete[] C_h;


	return 0;
}
